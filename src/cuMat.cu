#include "hip/hip_runtime.h"
﻿

#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../inc/common.h"
#include "cuda_app_def.h"
#include "mat.h"
#include "cuMat.h"

/*矩阵旋转180度*/
__global__ void cuMat2dRotate_180_kernel(float *matIn, float *matOut, unsigned int outW, unsigned int outH)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	if (!matIn || !matOut || !outW || !outH)
	{
		return;
	}
	int row = blockIdx.y*blockDim.y + threadIdx.y;  // X 对应矩阵row, Y对应举证col
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	matOut[j * outW + i] = matIn[i * outH+ j];

}

__global__ void addKernel(float *c, const float *a, const float *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

//矩阵加法的kernel
__global__ void cuMat2DAdd_kernel(float* _C, const float* _A, const float *_B, int n)
{

	//找出该线程所在的行列
	int row = (blockIdx.x*blockDim.x + threadIdx.x)/n;  // X 对应矩阵row, Y对应举证col
	int col = (blockIdx.x*blockDim.x + threadIdx.x)%n;
	if (row < n)
	{
		//线程Thread(row,col)负责计算C(row,col)
		_C[col + row*n] = _A[col + row*n] + _B[col + row*n];
	}
}

__global__ void cuMat2DRolate180_kernel(float* _B, const float* _A, int n)
{

	//找出该线程所在的行列
	int row = (blockIdx.x*blockDim.x + threadIdx.x) / n;  // X 对应矩阵row, Y对应举证col
	int col = (blockIdx.x*blockDim.x + threadIdx.x) % n;
	if (row < n)
	{
		//线程Thread(row,col)负责计算C(row,col)
		_B[col + row*n] = _A[(n-1-col) + (n-1-row)*n];
	}
}

__global__ void cuMat2DSubSum_kernel(float* inData, nSize inSize, float* mapMat, nSize mapSize, float* outData, int outSizeW)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	int r, c;
//	printf("x:%d, y:%d\n", blockDim.x, blockDim.y);
	outData[j * outSizeW + i] = (float)0.0;
	for (r = 0; r < mapSize.r; r++)
	{
		for (c = 0; c < mapSize.c; c++)
		{
			outData[j * outSizeW + i] += mapMat[r * mapSize.c + c] * inData[(j + r) * inSize.c + i + c];
		}
	}
}

__global__ void cuMat2dEdgeExpand_kernel(float *matIn, nSize matSize, float *matOut, int addc, int addr)
{
	int col, row;
	int out_c = matSize.c + 2 * addc;
	col = blockIdx.x*blockDim.x + threadIdx.x;
	row = blockIdx.y*blockDim.y + threadIdx.y;
//	printf("x:%d, y:%d\n", blockDim.x, blockDim.y);

	if (row < addr || row >= (matSize.r + addr) \
		|| col < addc || col >= (matSize.c + addc))
	{
		matOut[row * out_c + col] = (float)0.0;
	}		
	else
	{
		matOut[row * out_c + col] = matIn[(row - addr) * matSize.c + col - addc]; /* 复制原向量的数据 */
	}
}

__global__ void cuMat2dEdgeShrink_kernel(float *matIn, nSize matSize, float *matOut, int shrinkc, int shrinkr)
{
	int i, j;
	int w = matSize.c;
	int h = matSize.r;
	i = threadIdx.x;
	j = threadIdx.y;
	
	if ((j >= shrinkr) && (i >= shrinkc) && (j < (h - shrinkr)) && (i < (w - shrinkc)))
	{
		matOut[(j - shrinkr) * (w - 2 * shrinkc) + i - shrinkc] = matIn[j * w + i]; /* 复制原向量的数据 */
	}

}

void cuMat2dCorrelation_Valid(float *srcMat, nSize srcSize, float *mapMat, nSize mapSize, float *dstMat, nSize dstSize)
{
	int i, j, c, r;
	float *pTmpData = NULL;
	nSize exSize = { 0, 0 };
	int halfmapsizew;
	int halfmapsizeh;
	hipError_t cuRet = hipSuccess;
	hipError_t cudaStatus = hipSuccess;

	if (!srcMat || !mapMat || !dstMat)
	{
		PRT_ERR("param error !\n");
		return;
	}

	if (mapSize.r % 2 == 0 && mapSize.c % 2 == 0)/* 模板大小为偶数 */
	{
		halfmapsizew = (mapSize.c) / 2; /* 卷积模块的半瓣大小 */
		halfmapsizeh = (mapSize.r) / 2;
	}
	else
	{
		halfmapsizew = (mapSize.c - 1) / 2; /* 卷积模块的半瓣大小 */
		halfmapsizeh = (mapSize.r - 1) / 2;
	}

	/* 这里先默认进行full模式的操作，full模式的输出大小为inSize+(mapSize-1) */
	int outSizeW = srcSize.c + (mapSize.c - 1); /* 这里的输出扩大一部分 */
	int outSizeH = srcSize.r + (mapSize.r - 1);
	nSize outSize = { outSizeW, outSizeH };

	float *pOutDataDev = NULL;
	cuRet = hipMalloc((void**)&pOutDataDev, outSizeW*outSizeH*sizeof(float));
	RET_CHEAK_ZERO(cuRet);

	/* 这里先默认进行full模式的操作，full模式的输出大小为inSize+(mapSize-1) */
	/* 为了方便计算，将inputData扩大一圈 */
	exSize.c = srcSize.c + 2 * (mapSize.c - 1);
	exSize.r = srcSize.r + 2 * (mapSize.r - 1);


	float *pTmpDev = NULL;
	cuRet = hipMalloc((void**)&pTmpDev, exSize.c*exSize.r*sizeof(float));
	RET_CHEAK_ZERO(cuRet);

	dim3 blk;
	dim3 gid;
	gid.x = 2;
	gid.y = 2;
	gid.z = 1;
		 
	blk.x = exSize.c/2;
	blk.y = exSize.r/2;
	blk.z = 1;
//	printf("x,y,z: %d-%d-%d\n", blk.x, blk.y, blk.z);
	cuMat2dEdgeExpand_kernel << <gid, blk >> >(srcMat, srcSize, pTmpDev, mapSize.c - 1, mapSize.r - 1);
	CUDA_STS_CHECK(cudaStatus);
	hipDeviceSynchronize();

	blk.x = outSizeW;
	blk.y = outSizeH;
//	printf("x,y,z: %d-%d-%d\n", blk.x, blk.y, blk.z);
	cuMat2DSubSum_kernel << <1, blk >> >(pTmpDev, exSize, mapMat, mapSize, pOutDataDev, outSizeW);
	CUDA_STS_CHECK(cudaStatus);
	hipDeviceSynchronize();

	blk.x = outSizeW;
	blk.y = outSizeH;
	if (mapSize.r % 2 == 0 && mapSize.c % 2 == 0)/* 模板大小为偶数 */
	{
		cuMat2dEdgeShrink_kernel << <1, blk >> >(pOutDataDev, outSize, dstMat, halfmapsizew * 2-1, halfmapsizeh * 2-1);
	}
	else
	{
		cuMat2dEdgeShrink_kernel << <1, blk >> >(pOutDataDev, outSize, dstMat, halfmapsizew * 2, halfmapsizeh * 2);
	}
	CUDA_STS_CHECK(cudaStatus);
	hipDeviceSynchronize();
	hipFree(pTmpDev);
	hipFree(pOutDataDev);
}

