#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <math.h>
/* #include <random.h> */
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "cnn.h"
#include "cuMat.h"
#include "cuCnn.h"


void cuCnnDestroy(CNN_NET_STR *stCnnNet)
{
	int i = 0;
	hipError_t cuRet = hipSuccess;
	CNN_OUT_LAYER_L5 *pOutLayer = NULL;
	CHEAK_POINT_NULL(stCnnNet);


	pOutLayer = &stCnnNet->stOutL5;
	hipFree((void*)pOutLayer->vDev);
	hipFree((void*)pOutLayer->basicDev);
	hipFree((void*)pOutLayer->yDev);
	PRT("cnn memory released ok !\n");
}

/* CNN��ʼ������ */
void cuCnnSetUp(CNN_NET_STR *stCnnNet)
{
	int i = 0;
	hipError_t cuRet = hipSuccess;
	CNN_POOLING_LAYER_L2 *pPoolingLayer2 = NULL;
	CNN_POOLING_LAYER_L4 *pPoolingLayer4 = NULL;
	CNN_OUT_LAYER_L5 *pOutLayer = NULL;
	if (!stCnnNet)
	{
		PRT_ERR("param error\n");
	}

	cuRet = hipMalloc((void **)&stCnnNet->pInDataDev, CNN_LAYER1_IN_SIZE*CNN_LAYER1_IN_SIZE*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);

	stCnnNet->layerNum = CNN_LAYER_NUM;
	//########################################################################//
	stCnnNet->stCovL1.inChannels = CNN_LAYER1_IN_CHANNEL_NUM;
	stCnnNet->stCovL1.outChannels = CNN_LAYER1_OUT_CHANNEL_NUM;
	stCnnNet->stCovL1.isFullConnect = TRUE;
	stCnnNet->stCovL1.inputWidth = CNN_LAYER1_IN_SIZE;
	stCnnNet->stCovL1.inputHeight = CNN_LAYER1_IN_SIZE;
	stCnnNet->stCovL1.mapSize = CNN_LAYER1_MAP_SIZE;
	for (i = 0; i < CNN_LAYER1_OUT_CHANNEL_NUM; i++)
	{
		cuRet = hipMalloc((void **)&stCnnNet->stCovL1.vDev[i], CNN_LAYER1_OUT_SIZE*CNN_LAYER1_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
		cuRet = hipMalloc((void **)&stCnnNet->stCovL1.yDev[i], CNN_LAYER1_OUT_SIZE*CNN_LAYER1_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
		cuRet = hipMalloc((void **)&stCnnNet->stCovL1.dDev[i], CNN_LAYER1_OUT_SIZE*CNN_LAYER1_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
	}
	cuRet = hipMalloc((void **)&stCnnNet->stCovL1.mapOutDev, CNN_LAYER1_OUT_SIZE*CNN_LAYER1_OUT_SIZE*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void**)&stCnnNet->stCovL1.basicDev, CNN_LAYER1_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	//########################################################################//
	pPoolingLayer2 = &stCnnNet->stPoolL2;
	stCnnNet->stPoolL2.inChannels = CNN_LAYER2_IN_CHANNEL_NUM;
	stCnnNet->stPoolL2.outChannels = CNN_LAYER2_IN_CHANNEL_NUM;
	stCnnNet->stPoolL2.inputWidth = CNN_LAYER1_OUT_SIZE;
	stCnnNet->stPoolL2.inputHeight = CNN_LAYER1_OUT_SIZE;
	stCnnNet->stPoolL2.mapSize = 2;
	stCnnNet->stPoolL2.poolType = CNN_AVE_POOL;
	cuRet = hipMalloc((void **)&pPoolingLayer2->basicDev, CNN_LAYER2_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	for (i = 0; i < CNN_LAYER2_OUT_CHANNEL_NUM; i++)
	{
		cuRet = hipMalloc((void **)&pPoolingLayer2->yDev[i], CNN_LAYER2_OUT_SIZE*CNN_LAYER2_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
	}
	cuRet = hipMalloc((void **)&pPoolingLayer2->dDev, CNN_LAYER2_OUT_SIZE*CNN_LAYER2_OUT_SIZE*CNN_LAYER2_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	//########################################################################//
	stCnnNet->stCovL3.inChannels = CNN_LAYER3_IN_CHANNEL_NUM;
	stCnnNet->stCovL3.outChannels = CNN_LAYER3_OUT_CHANNEL_NUM;
	stCnnNet->stCovL3.isFullConnect = TRUE;
	stCnnNet->stCovL3.inputWidth = CNN_LAYER3_IN_SIZE;
	stCnnNet->stCovL3.inputHeight = CNN_LAYER3_IN_SIZE;
	stCnnNet->stCovL3.mapSize = CNN_LAYER3_MAP_SIZE;
	for (i = 0; i < CNN_LAYER3_OUT_CHANNEL_NUM; i++)
	{
		cuRet = hipMalloc((void **)&stCnnNet->stCovL3.vDev[i], CNN_LAYER3_OUT_SIZE*CNN_LAYER3_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
		cuRet = hipMalloc((void **)&stCnnNet->stCovL3.yDev[i], CNN_LAYER3_OUT_SIZE*CNN_LAYER3_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
		cuRet = hipMalloc((void **)&stCnnNet->stCovL3.dDev[i], CNN_LAYER3_OUT_SIZE*CNN_LAYER3_OUT_SIZE*sizeof(FLOAT));
		RET_CHEAK_ZERO(cuRet);
	}
	cuRet = hipMalloc((void **)&stCnnNet->stCovL3.mapOutDev, CNN_LAYER3_OUT_SIZE*CNN_LAYER3_OUT_SIZE*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void**)&stCnnNet->stCovL3.basicDev, CNN_LAYER3_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	//########################################################################//
	pPoolingLayer4 = &stCnnNet->stPoolL4;
	stCnnNet->stPoolL4.inChannels = CNN_LAYER4_IN_CHANNEL_NUM;
	stCnnNet->stPoolL4.outChannels = CNN_LAYER4_IN_CHANNEL_NUM;
	stCnnNet->stPoolL4.inputWidth = CNN_LAYER3_OUT_SIZE;
	stCnnNet->stPoolL4.inputHeight = CNN_LAYER3_OUT_SIZE;
	stCnnNet->stPoolL4.mapSize = 2;
	stCnnNet->stPoolL4.poolType = CNN_AVE_POOL;


	cuRet = hipMalloc((void **)&pPoolingLayer4->basicDev, CNN_LAYER4_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void **)&pPoolingLayer4->yDev, CNN_LAYER4_OUT_SIZE*CNN_LAYER4_OUT_SIZE*CNN_LAYER4_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void **)&pPoolingLayer4->dDev, CNN_LAYER4_OUT_SIZE*CNN_LAYER4_OUT_SIZE*CNN_LAYER4_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);

	//########################################################################//
	stCnnNet->stOutL5.inputNum = CNN_LAYER5_IN_DATA_NUM;
	stCnnNet->stOutL5.outputNum = CNN_LAYER5_OUT_CHANNEL_NUM;
	stCnnNet->stOutL5.isFullConnect = TRUE;
	pOutLayer = &stCnnNet->stOutL5;
	cuRet = hipMalloc((void**)&pOutLayer->inDev, CNN_LAYER5_IN_DATA_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void**)&pOutLayer->basicDev, CNN_LAYER5_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void**)&pOutLayer->vDev, CNN_LAYER5_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void**)&pOutLayer->yDev, CNN_LAYER5_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
	cuRet = hipMalloc((void**)&pOutLayer->dDev, CNN_LAYER5_OUT_CHANNEL_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);

	cuRet = hipMalloc((void**)&pOutLayer->wDev, CNN_LAYER5_OUT_CHANNEL_NUM*CNN_LAYER5_IN_DATA_NUM*sizeof(FLOAT));
	RET_CHEAK_ZERO(cuRet);
}

/*******************************************************************************
Function:		cuSigmaActiveFun
Description:
Input:
Output:		N/A
Return:		0:			Successful
ohters:		Failed
*******************************************************************************/
__global__ void cuSigmaActiveFun_kernel(float * input, float *bas, float *output) /* sigma����� */
{	
	int i = threadIdx.x;
	output[i] = (float)1.0 / ((float)(1.0 + exp(-(input[i] + (float)*bas))));
}

__global__ void cuSigmaActiveL5_kernel(float * input, float *bas, float *output) /* sigma����� */
{
	int i = threadIdx.x;
	output[i] = (float)1.0 / ((float)(1.0 + exp(-(input[i] + bas[i]))));
}

__global__ void cuAverageL2_kernel(float * input, float *output)
{
	FLOAT sum = 0.0;
	int w = CNN_LAYER2_OUT_SIZE;
	int mapSize = 2;
	int m, n;
	int i = threadIdx.x;
	int j = threadIdx.y;

	for (m = j * mapSize; m < (j+1) * mapSize; m++)
	{
		for (n = i * mapSize; n < (i+1) * mapSize; n++)
		{
			sum += input[m * w *mapSize + n];
		}
	}

	output[i+j*w] = sum / (float)(mapSize * mapSize * 1.0);
}
/*******************************************************************************
Function:		cuPoolingAverage
Description:  ��ƽ��ֵ 
Input:
Output:		N/A
Return:		0:			Successful
ohters:		Failed
*******************************************************************************/
void cuL2PoolingAverage(float *output, nSize outputSize, float *input, nSize inputSize, int mapSize)
{
	int i, j, m, n;
	float sum = 0.0;
	int outputW = 0;
	int outputH = 0;
	CHEAK_POINT_NULL(input);
	CHEAK_POINT_NULL(output);
	CHEAK_VALUE_ZERO(mapSize);

	outputW = inputSize.c / mapSize;
	outputH = inputSize.r / mapSize;

	if (outputSize.c != outputW || outputSize.r != outputH)
	{
		PRT("ERROR: output size is wrong!!");
		return;
	}

	FLOAT * data1 = NULL;
	FLOAT * data2 = NULL;

	hipMalloc((void**)&data1, (inputSize.c*inputSize.r)*sizeof(FLOAT));
	hipMalloc((void**)&data2, (outputW*outputH)*sizeof(FLOAT));

	hipMemcpy((void*)(data1), (void*)(input), (inputSize.c*inputSize.r)*sizeof(FLOAT), hipMemcpyHostToDevice);
	dim3 dimBlock(CNN_LAYER2_OUT_SIZE, CNN_LAYER2_OUT_SIZE);
	cuAverageL2_kernel << <1, dimBlock >> >(data1, data2);
	hipDeviceSynchronize();

	hipMemcpy((void*)(output), (void*)(data2), (CNN_LAYER2_OUT_SIZE*CNN_LAYER2_OUT_SIZE)*sizeof(FLOAT), hipMemcpyDeviceToHost);
	hipFree((void*)data1);
	hipFree((void*)data2);
}

__global__ void cuAverageL4_kernel(float * input, float *output)
{
	FLOAT sum = 0.0;
	int w = CNN_LAYER4_OUT_SIZE;
	int mapSize = 2;
	int m, n;
	int i = threadIdx.x;
	int j = threadIdx.y;

	for (m = j * mapSize; m < (j + 1) * mapSize; m++)
	{
		for (n = i * mapSize; n < (i + 1) * mapSize; n++)
		{
			sum += input[m * w *mapSize + n];
		}
	}

	output[i + j*w] = sum / (float)(mapSize * mapSize * 1.0);
}

void cuL4PoolingAverage(float *output, nSize outputSize, float *input, nSize inputSize, int mapSize)
{
	int i, j, m, n;
	float sum = 0.0;
	int outputW = 0;
	int outputH = 0;
	CHEAK_POINT_NULL(input);
	CHEAK_POINT_NULL(output);
	CHEAK_VALUE_ZERO(mapSize);

	outputW = inputSize.c / mapSize;
	outputH = inputSize.r / mapSize;

	if (outputSize.c != outputW || outputSize.r != outputH)
	{
		PRT("ERROR: output size is wrong!!");
		return;
	}

	FLOAT * data1 = NULL;
	FLOAT * data2 = NULL;

	hipMalloc((void**)&data1, (inputSize.c*inputSize.r)*sizeof(FLOAT));
	hipMalloc((void**)&data2, (outputW*outputH)*sizeof(FLOAT));

	hipMemcpy((void*)(data1), (void*)(input), (inputSize.c*inputSize.r)*sizeof(FLOAT), hipMemcpyHostToDevice);
	dim3 dimBlock(CNN_LAYER4_OUT_SIZE, CNN_LAYER4_OUT_SIZE);
	cuAverageL4_kernel << <1, dimBlock >> >(data1, data2);
	hipDeviceSynchronize();

	hipMemcpy((void*)(output), (void*)(data2), (CNN_LAYER4_OUT_SIZE*CNN_LAYER4_OUT_SIZE)*sizeof(FLOAT), hipMemcpyDeviceToHost);
	hipFree((void*)data1);
	hipFree((void*)data2);
}

/*******************************************************************************
Function:		cuCnnTrainProc
Description:
Input:
Output:		N/A
Return:		0:			Successful
ohters:		Failed
*******************************************************************************/
void cuCnnTrainProc(CNN_NET_STR *pCnnNet, MinstImgArr *inputData, MinstLabelArr *outputData, CNNOpts opts, int trainNum)
{
	int e = 0;
	int i = 0;
	int j = 0;
	int k = 0;
	int n = 0;
	float iee = 0.0;
	char fileName[128] = { '\0' };
	FLOAT imgDataFloat[CNN_LAYER1_IN_SIZE][CNN_LAYER1_IN_SIZE] = { { 0.0 } };/* �Ҷ�ͼ */

	if (!pCnnNet || !inputData || !outputData)
	{
		return;
	}

	for (e = 0; e < opts.numepochs; e++)
	{
		for (n = 0; n < trainNum; n++)
		{
			PRT("numepochs:%d, img:%d \n", e, n);
			for (j = 0; j < CNN_LAYER1_IN_SIZE; j++)
			{
				for (k = 0; k < CNN_LAYER1_IN_SIZE; k++)
				{
					imgDataFloat[j][k] = inputData->ImgPtr[n].ImgData[j][k];
				}
			}

			CnnForwardPass(pCnnNet, (FLOAT *)imgDataFloat);  /* ǰ�򴫲���������Ҫ����� */

			CnnBackPass(pCnnNet, outputData->LabelPtr[n].LabelData); /* ���򴫲���������Ҫ�������Ԫ������ݶ� */

			/*            sprintf(fileName, "/mnt/hgfs/share/cnnDemo/PicTrans/CnnData1/%d.cnn", n); */
			/*            SaveCnnMidData(pCnnNet, fileName, inputData->ImgPtr[n].ImgData); */

			CnnApplyGrads(pCnnNet, opts, (FLOAT *)imgDataFloat); /* ����Ȩ�� */

			CnnParamClear(pCnnNet);
			/* ���㲢����������� */
			iee = 0.0;
			for (i = 0; i < pCnnNet->stOutL5.outputNum; i++)
			{
				iee = iee + pCnnNet->e[i] * pCnnNet->e[i];
			}

			if (n == 0)
				pCnnNet->L[n] = iee / (float)2.0;
			else
				pCnnNet->L[n] = pCnnNet->L[n - 1] * 0.99 + 0.01 * iee / (float)2.0;
		}
	}
}

/* ����InputData��ͼ�����ݣ�inputData[r][c],r��c�У��������Ȩ��ģ����һ�µ� */
void cuCnnff(CNN *cnn, float **inputData)
{
	int i, j, r, c;

	if (!cnn || !inputData)
	{
		return;
	}

	int outSizeW = cnn->S2->inputWidth;
	int outSizeH = cnn->S2->inputHeight;
	/* ��һ��Ĵ��� */

	/* ��һ��������� */
	nSize mapSize = { cnn->C1->mapSize, cnn->C1->mapSize };
	nSize inSize = { cnn->C1->inputWidth, cnn->C1->inputHeight };
	nSize outSize = { cnn->S2->inputWidth, cnn->S2->inputHeight };
	PRT("C1: insize[%d-%d], mapsize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, mapSize.c, mapSize.r, outSize.c, outSize.r);
	for (i = 0; i < (cnn->C1->outChannels); i++)
	{
		for (j = 0; j < (cnn->C1->inChannels); j++)
		{
			float **mapout = cov(cnn->C1->mapData[j][i], mapSize, inputData, inSize, COV_VALID);
			addmat(cnn->C1->v[i], cnn->C1->v[i], outSize, mapout, outSize);
			for (r = 0; r < outSize.r; r++)
			{
				free(mapout[r]);
			}

			free(mapout);
		}

		for (r = 0; r < outSize.r; r++)
		{
			for (c = 0; c < outSize.c; c++)
				cnn->C1->y[i][r][c] = activation_Sigma(cnn->C1->v[i][r][c], cnn->C1->basicData[i]);
		}
	}

	CovLayerPrint(cnn->C1);

	/* �ڶ�����������S2�������� */
	outSize.c = cnn->C3->inputWidth;
	outSize.r = cnn->C3->inputHeight;
	inSize.c = cnn->S2->inputWidth;
	inSize.r = cnn->S2->inputHeight;
	PRT("S2: insize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, outSize.c, outSize.r);
	for (i = 0; i < (cnn->S2->outChannels); i++)
	{
		if (cnn->S2->poolType == CNN_AVE_POOL)
			avgPooling(cnn->S2->y[i], outSize, cnn->C1->y[i], inSize, cnn->S2->mapSize);
	}

	PoolLayerPrint(cnn->S2);
	/* �������������,������ȫ���� */
	outSize.c = cnn->S4->inputWidth;
	outSize.r = cnn->S4->inputHeight;
	inSize.c = cnn->C3->inputWidth;
	inSize.r = cnn->C3->inputHeight;
	mapSize.c = cnn->C3->mapSize;
	mapSize.r = cnn->C3->mapSize;
	PRT("C3: insize[%d-%d], mapsize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, mapSize.c, mapSize.r, outSize.c, outSize.r);
	for (i = 0; i < (cnn->C3->outChannels); i++)
	{
		for (j = 0; j < (cnn->C3->inChannels); j++)
		{
			float **mapout = cov(cnn->C3->mapData[j][i], mapSize, cnn->S2->y[j], inSize, COV_VALID);
			addmat(cnn->C3->v[i], cnn->C3->v[i], outSize, mapout, outSize);
			for (r = 0; r < outSize.r; r++)
				free(mapout[r]);

			free(mapout);
		}

		for (r = 0; r < outSize.r; r++)
		for (c = 0; c < outSize.c; c++)
			cnn->C3->y[i][r][c] = activation_Sigma(cnn->C3->v[i][r][c], cnn->C3->basicData[i]);
	}

	CovLayerPrint(cnn->C3);

	/* ���Ĳ��������� */
	inSize.c = cnn->S4->inputWidth;
	inSize.r = cnn->S4->inputHeight;
	outSize.c = inSize.c / cnn->S4->mapSize;
	outSize.r = inSize.r / cnn->S4->mapSize;
	PRT("S4: insize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, outSize.c, outSize.r);
	for (i = 0; i < (cnn->S4->outChannels); i++)
	{
		if (cnn->S4->poolType == AvePool)
			avgPooling(cnn->S4->y[i], outSize, cnn->C3->y[i], inSize, cnn->S4->mapSize);
	}

	PoolLayerPrint(cnn->S4);
	/* �����O5�Ĵ��� */
	/* ������Ҫ��ǰ��Ķ�ά���չ����һά���� */
	float *O5inData = (float *)malloc((cnn->O5->inputNum) * sizeof(float));
	for (i = 0; i < (cnn->S4->outChannels); i++)
	{
		for (r = 0; r < outSize.r; r++)
		{
			for (c = 0; c < outSize.c; c++)
				O5inData[i * outSize.r * outSize.c + r * outSize.c + c] = cnn->S4->y[i][r][c];
		}
	}

	nSize cnnL5nSize = { cnn->O5->inputNum, cnn->O5->outputNum };/* 192-10 */
	nnff(cnn->O5->v, O5inData, cnn->O5->wData, cnn->O5->basicData, cnnL5nSize);
	for (i = 0; i < cnn->O5->outputNum; i++)
	{
		cnn->O5->y[i] = activation_Sigma(cnn->O5->v[i], cnn->O5->basicData[i]);
	}

	free(O5inData);
	OutLayerPrint(cnn->O5);
}

/*******************************************************************************
Function:		cuNn2f_kernel
Description:
Input:
Output:		N/A
Return:		0:			Successful
ohters:		Failed
*******************************************************************************/
__global__ void cuNn2f_kernel(float *output, float *input, float *wdata, float *bas, nSize nnSize)
{
	int w = nnSize.c;
	int h = nnSize.r;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = 0, ty = 0;


	if (!output || !input || !wdata || !bas)
	{
		PRT_ERR("param error\n");
		return;
	}
	tx = threadIdx.x;
	ty = threadIdx.y;

	//input 1*192->192*1
	// w 10*192
	// output 10*1

	__shared__ float tmp[192];

	tmp[tx] = wdata[192 * bx + tx] * input[tx];
	__syncthreads();

	if (0 == tx)
	{
		output[bx] = 0.0;
		for (int i = 0; i < 192; i++)
		{
			output[bx] += tmp[i];
		}
		output[bx] += bas[bx];
	}
	__syncthreads();
}

/* ����InputData��ͼ�����ݣ�inputData[r][c],r��c�У��������Ȩ��ģ����һ�µ� */
void cuCnnForwardPass(CNN_NET_STR *pCnnNet, float *inputData)
{
	int i, j, r, c;
	int kr, kc;
	int idx = 0;
	FILE *fp = NULL;
	unsigned char imgName[128] = { '\0' };
	unsigned char imgData[1024] = { 0 };
	nSize dstSize = { 0, 0 };
	FLOAT mapDataRotL1[CNN_LAYER1_MAP_SIZE][CNN_LAYER1_MAP_SIZE] = { { 0.0 } };
	FLOAT mapDataRotL3[CNN_LAYER3_MAP_SIZE][CNN_LAYER3_MAP_SIZE] = { { 0.0 } };
	hipError_t cuRet = hipSuccess;
	CNN_OUT_LAYER_L5 *pOutLayer = NULL;
	if (!pCnnNet || !inputData)
	{
		return;
	}

	int outSizeW = pCnnNet->stPoolL2.inputWidth;
	int outSizeH = pCnnNet->stPoolL2.inputHeight;
	/* ��һ��Ĵ��� */

	/* ��һ��������� */
	nSize mapSize = { pCnnNet->stCovL1.mapSize, pCnnNet->stCovL1.mapSize };
	nSize inSize = { pCnnNet->stCovL1.inputWidth, pCnnNet->stCovL1.inputHeight };
	nSize outSize = { pCnnNet->stPoolL2.inputWidth, pCnnNet->stPoolL2.inputHeight };
	/*    PRT("L1: insize[%d-%d], mapsize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, mapSize.c, mapSize.r, outSize.c, outSize.r); */
	/*	PRT("L1:inChannels:%d, outChannels:%d, mapSize:%d\n", pCnnNet->stCovL1.inChannels, pCnnNet->stCovL1.outChannels, pCnnNet->stCovL1.mapSize); */
	hipMemcpy((void*)(pCnnNet->stCovL1.basicDev), (void*)(pCnnNet->stCovL1.basicData), CNN_LAYER1_OUT_CHANNEL_NUM * sizeof(float), hipMemcpyHostToDevice);
	for (i = 0; i < (pCnnNet->stCovL1.outChannels); i++)
	{
		for (j = 0; j < (pCnnNet->stCovL1.inChannels); j++)
		{

			/* ����COV_VALID���;�� */
			if (mapSize.r % 2 == 0 && mapSize.c % 2 == 0)/* ż����һ�� r cһ�� */
			{
				dstSize.c = inSize.c + 3 - mapSize.c;
				dstSize.r = inSize.r + 3 - mapSize.r;
			}
			else/* ���� */
			{
				dstSize.c = inSize.c + 1 - mapSize.c;
				dstSize.r = inSize.r + 1 - mapSize.r;
			}

			dstSize.c = CNN_LAYER1_OUT_SIZE;
			dstSize.r = CNN_LAYER1_OUT_SIZE;
			/*			Mat2dRotate_180(pCnnNet->stCovL1.mapData[j][i], mapDataRotL1, mapSize); */
#if 0
			/* ӳ�������ת���������Ϊ��ؼ��� */
			for (kr = 0; kr < CNN_LAYER1_MAP_SIZE; kr++)
			{
				for (kc = 0; kc < CNN_LAYER1_MAP_SIZE; kc++)
				{
					mapDataRotL1[kr][kc] = pCnnNet->stCovL1.mapData[j][i][CNN_LAYER1_MAP_SIZE - kr - 1][CNN_LAYER1_MAP_SIZE - kc - 1];
				}
			}
			memset(pCnnNet->stCovL1.mapOut, 0, CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE * sizeof(float));
			Mat2dCorrelation_Valid(inputData, inSize, (FLOAT *)mapDataRotL1, mapSize, (FLOAT *)pCnnNet->stCovL1.mapOut, dstSize);
			Mat2D_Add((FLOAT *)pCnnNet->stCovL1.v[i], (FLOAT *)pCnnNet->stCovL1.v[i], (FLOAT *)pCnnNet->stCovL1.mapOut, dstSize);
#else
			float * mapDataRotL1Dev = NULL;
			float * mapDataDev = NULL;
			cuRet = hipMalloc((void**)&mapDataRotL1Dev, CNN_LAYER1_MAP_SIZE*CNN_LAYER1_MAP_SIZE*sizeof(FLOAT));
			RET_CHEAK_ZERO(cuRet);
			cuRet = hipMalloc((void **)&mapDataDev, CNN_LAYER1_MAP_SIZE*CNN_LAYER1_MAP_SIZE*sizeof(FLOAT));
			RET_CHEAK_ZERO(cuRet);

			hipMemcpy((void*)mapDataDev, (void*)pCnnNet->stCovL1.mapData[j][i], CNN_LAYER1_MAP_SIZE*CNN_LAYER1_MAP_SIZE*sizeof(FLOAT), hipMemcpyHostToDevice);
			cuMat2DRolate180_kernel << <CNN_LAYER1_OUT_SIZE, CNN_LAYER1_OUT_SIZE >> >(mapDataRotL1Dev, mapDataDev, CNN_LAYER1_MAP_SIZE);
			hipDeviceSynchronize();
			hipMemcpy((void*)mapDataRotL1, (void*)mapDataRotL1Dev, CNN_LAYER1_MAP_SIZE*CNN_LAYER1_MAP_SIZE*sizeof(FLOAT), hipMemcpyDeviceToHost);

			memset(pCnnNet->stCovL1.mapOut, 0, CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE * sizeof(float));
			hipMemset(pCnnNet->stCovL1.mapOutDev, 0, CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE*sizeof(FLOAT));

			hipMemcpy((void*)pCnnNet->pInDataDev, inputData, CNN_LAYER1_IN_SIZE*CNN_LAYER1_IN_SIZE*sizeof(FLOAT), hipMemcpyHostToDevice);
			cuMat2dCorrelation_Valid(pCnnNet->pInDataDev, inSize, (FLOAT *)mapDataRotL1Dev, mapSize, (FLOAT *)pCnnNet->stCovL1.mapOutDev, dstSize);
			hipMemcpy((void*)pCnnNet->stCovL1.mapOut, pCnnNet->stCovL1.mapOutDev, CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE*sizeof(FLOAT), hipMemcpyDeviceToHost);
			cuRet = hipFree(mapDataDev);
			RET_CHEAK_ZERO(cuRet);
			hipFree(mapDataRotL1Dev);

			hipMemcpy((void*)(pCnnNet->stCovL1.vDev[i]), (void*)(pCnnNet->stCovL1.v[i]), CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
			cuMat2DAdd_kernel << <CNN_LAYER1_OUT_SIZE, CNN_LAYER1_OUT_SIZE>> >(pCnnNet->stCovL1.vDev[i], pCnnNet->stCovL1.vDev[i], pCnnNet->stCovL1.mapOutDev, CNN_LAYER1_OUT_SIZE);
			hipDeviceSynchronize();
			hipMemcpy((void*)(pCnnNet->stCovL1.v[i]), (void*)(pCnnNet->stCovL1.vDev[i]), CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);
#endif
		}
#if 0
		for (r = 0; r < outSize.r; r++)
		{
			for (c = 0; c < outSize.c; c++)
			{
				pCnnNet->stCovL1.y[i][r][c] = SigmaActiveFun(pCnnNet->stCovL1.v[i][r][c], pCnnNet->stCovL1.basicData[i]);
			}
		}
#else		
		cuSigmaActiveFun_kernel << <1, CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE>> >(pCnnNet->stCovL1.vDev[i], (float*)&pCnnNet->stCovL1.basicDev[i], pCnnNet->stCovL1.yDev[i]);
		hipDeviceSynchronize();
		hipMemcpy(pCnnNet->stCovL1.y[i], pCnnNet->stCovL1.yDev[i], CNN_LAYER1_OUT_SIZE * CNN_LAYER1_OUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);
#endif
	}

	/*
	PRT("L1: output data ");
	for (i = 0; i < (pCnnNet->stCovL1.outChannels); i++){
	idx = 0;
	sprintf(imgName, "/mnt/hgfs/share/cnnDemo/output/imgL1_%d.yuv", i);
	fp = fopen(imgName, "wb");
	for (r = 0; r < CNN_LAYER1_OUT_SIZE; r++){
	for (c = 0; c < CNN_LAYER1_OUT_SIZE; c++){
	PRT("%f ", pCnnNet->stCovL1.y[i][r][c]);
	imgData[idx] = (unsigned char)(pCnnNet->stCovL1.y[i][r][c] * 255);
	idx++;
	}
	PRT("\n");
	}
	PRT("\n");

	fwrite(imgData, CNN_LAYER1_OUT_SIZE*CNN_LAYER1_OUT_SIZE, 1 ,fp);
	fclose(fp);
	}
	*/

	/* �ڶ�����������S2�������� */
	inSize.c = pCnnNet->stPoolL2.inputWidth;
	inSize.r = pCnnNet->stPoolL2.inputHeight;
	outSize.c = pCnnNet->stCovL3.inputWidth;
	outSize.r = pCnnNet->stCovL3.inputHeight;
	/*    PRT("S2: insize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, outSize.c, outSize.r); */

	for (i = 0; i < (pCnnNet->stPoolL2.outChannels); i++)
	{
		if (pCnnNet->stPoolL2.poolType == CNN_AVE_POOL)
		{
#if 0
			PoolingAverage((FLOAT *)pCnnNet->stPoolL2.y[i], outSize, (FLOAT *)pCnnNet->stCovL1.y[i], inSize, pCnnNet->stPoolL2.mapSize);
#else
			cuL2PoolingAverage((FLOAT *)pCnnNet->stPoolL2.y[i], outSize, (FLOAT *)pCnnNet->stCovL1.y[i], inSize, pCnnNet->stPoolL2.mapSize);
#endif
		}
	}

	/*
	PRT("L2: output data \n");
	for (i = 0; i < (pCnnNet->stPoolL2.outChannels); i++){
	for (r = 0; r < CNN_LAYER2_OUT_SIZE; r++){
	for (c = 0; c < CNN_LAYER2_OUT_SIZE; c++){
	PRT("%f ", pCnnNet->stPoolL2.y[i][r][c]);
	}
	PRT("\n");
	}
	PRT("\n");
	}
	*/

	/* �������������,������ȫ���� */
	inSize.c = pCnnNet->stCovL3.inputWidth;
	inSize.r = pCnnNet->stCovL3.inputHeight;
	mapSize.c = pCnnNet->stCovL3.mapSize;
	mapSize.r = pCnnNet->stCovL3.mapSize;
	outSize.c = pCnnNet->stPoolL4.inputWidth;
	outSize.r = pCnnNet->stPoolL4.inputHeight;
	/*    PRT("L3: insize[%d-%d], mapsize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, mapSize.c, mapSize.r, outSize.c, outSize.r); */
	/*	PRT("L3:inChannels:%d, outChannels:%d, mapSize:%d\n", pCnnNet->stCovL3.inChannels, pCnnNet->stCovL3.outChannels, pCnnNet->stCovL3.mapSize); */
	hipMemcpy((void*)(pCnnNet->stCovL3.basicDev), (void*)(pCnnNet->stCovL3.basicData), CNN_LAYER3_OUT_CHANNEL_NUM * sizeof(float), hipMemcpyHostToDevice);
	for (i = 0; i < (pCnnNet->stCovL3.outChannels); i++)
	{
		for (j = 0; j < (pCnnNet->stCovL3.inChannels); j++)
		{
			/* ����COV_VALID���;�� */
			if (mapSize.r % 2 == 0 && mapSize.c % 2 == 0)/* ż����һ�� r cһ�� */
			{
				dstSize.c = inSize.c + 3 - mapSize.c;
				dstSize.r = inSize.r + 3 - mapSize.r;
			}
			else/* ���� */
			{
				dstSize.c = inSize.c + 1 - mapSize.c;
				dstSize.r = inSize.r + 1 - mapSize.r;
			}

			dstSize.c = CNN_LAYER3_OUT_SIZE;
			dstSize.r = CNN_LAYER3_OUT_SIZE;
#if 0
			/*			Mat2dRotate_180(pCnnNet->stCovL3.mapData[j][i], mapDataRotL3, mapSize); */
			for (kr = 0; kr < CNN_LAYER3_MAP_SIZE; kr++)
			{
				for (kc = 0; kc < CNN_LAYER3_MAP_SIZE; kc++)
				{
					mapDataRotL3[kr][kc] = pCnnNet->stCovL3.mapData[j][i][CNN_LAYER3_MAP_SIZE - kr - 1][CNN_LAYER3_MAP_SIZE - kc - 1];
				}
			}
			memset(pCnnNet->stCovL3.mapOut, 0, CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float));
			Mat2dCorrelation_Valid((FLOAT *)pCnnNet->stPoolL2.y[j], inSize, (FLOAT *)mapDataRotL3, mapSize, (FLOAT *)pCnnNet->stCovL3.mapOut, dstSize);
			Mat2D_Add((FLOAT *)pCnnNet->stCovL3.v[i], (FLOAT *)pCnnNet->stCovL3.v[i], (FLOAT *)pCnnNet->stCovL3.mapOut, outSize);
#else
			float * mapDataRotL3Dev = NULL;
			float * mapDataL3Dev = NULL;
			cuRet = hipMalloc((void**)&mapDataRotL3Dev, CNN_LAYER3_MAP_SIZE*CNN_LAYER3_MAP_SIZE*sizeof(FLOAT));
			RET_CHEAK_ZERO(cuRet);
			cuRet = hipMalloc((void **)&mapDataL3Dev, CNN_LAYER3_MAP_SIZE*CNN_LAYER3_MAP_SIZE*sizeof(FLOAT));
			RET_CHEAK_ZERO(cuRet);

			hipMemcpy((void*)mapDataL3Dev, (void*)pCnnNet->stCovL3.mapData[j][i], CNN_LAYER3_MAP_SIZE*CNN_LAYER3_MAP_SIZE*sizeof(FLOAT), hipMemcpyHostToDevice);
			cuMat2DRolate180_kernel << <CNN_LAYER3_OUT_SIZE, CNN_LAYER3_OUT_SIZE >> >(mapDataRotL3Dev, mapDataL3Dev, CNN_LAYER3_MAP_SIZE);
			hipDeviceSynchronize();
			hipMemcpy((void*)mapDataRotL3, (void*)mapDataRotL3Dev, CNN_LAYER3_MAP_SIZE*CNN_LAYER3_MAP_SIZE*sizeof(FLOAT), hipMemcpyDeviceToHost);

			memset(pCnnNet->stCovL3.mapOut, 0, CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float));
			hipMemset(pCnnNet->stCovL3.mapOutDev, 0, CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE*sizeof(FLOAT));

			hipMemcpy((void*)pCnnNet->stPoolL2.yDev[j], pCnnNet->stPoolL2.y[j], CNN_LAYER3_IN_SIZE*CNN_LAYER3_IN_SIZE*sizeof(FLOAT), hipMemcpyHostToDevice);
			cuMat2dCorrelation_Valid(pCnnNet->stPoolL2.yDev[j], inSize, (FLOAT *)mapDataRotL3Dev, mapSize, (FLOAT *)pCnnNet->stCovL3.mapOutDev, dstSize);
			hipMemcpy((void*)pCnnNet->stCovL3.mapOut, pCnnNet->stCovL3.mapOutDev, CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE*sizeof(FLOAT), hipMemcpyDeviceToHost);

			hipFree(mapDataL3Dev);
			hipFree(mapDataRotL3Dev);

			hipMemcpy((void*)(pCnnNet->stCovL3.vDev[i]), (void*)(pCnnNet->stCovL3.v[i]), CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
			cuMat2DAdd_kernel << <CNN_LAYER3_OUT_SIZE, CNN_LAYER3_OUT_SIZE >> >(pCnnNet->stCovL3.vDev[i], pCnnNet->stCovL3.vDev[i], pCnnNet->stCovL3.mapOutDev, CNN_LAYER3_OUT_SIZE);
			hipDeviceSynchronize();
			hipMemcpy((void*)(pCnnNet->stCovL3.v[i]), (void*)(pCnnNet->stCovL3.vDev[i]), CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

			/*
			hipMemcpy((void*)(pCnnNet->stCovL3.vDev[i]), (void*)(pCnnNet->stCovL3.v[i]), CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy((void*)(pCnnNet->stCovL3.mapOutDev), (void*)(pCnnNet->stCovL3.mapOut), CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float), hipMemcpyHostToDevice);
			cuMat2DAdd_kernel << <CNN_LAYER3_OUT_SIZE, CNN_LAYER3_OUT_SIZE >> >(pCnnNet->stCovL3.vDev[i], pCnnNet->stCovL3.vDev[i], pCnnNet->stCovL3.mapOutDev, CNN_LAYER3_OUT_SIZE);
			hipDeviceSynchronize();
			hipMemcpy((void*)(pCnnNet->stCovL3.v[i]), (void*)(pCnnNet->stCovL3.vDev[i]), CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);
			*/
#endif
		}
#if 0
		for (r = 0; r < outSize.r; r++)
		{
			for (c = 0; c < outSize.c; c++)
			{
				pCnnNet->stCovL3.y[i][r][c] = SigmaActiveFun(pCnnNet->stCovL3.v[i][r][c], pCnnNet->stCovL3.basicData[i]);
			}
		}
#else		
		cuSigmaActiveFun_kernel << <1, CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE >> >(pCnnNet->stCovL3.vDev[i], (float*)&pCnnNet->stCovL3.basicDev[i], pCnnNet->stCovL3.yDev[i]);
		hipDeviceSynchronize();
		hipMemcpy(pCnnNet->stCovL3.y[i], pCnnNet->stCovL3.yDev[i], CNN_LAYER3_OUT_SIZE * CNN_LAYER3_OUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);
#endif
	}

	/*
	PRT("L3: output data \n");
	for (i = 0; i < (pCnnNet->stCovL3.outChannels); i++){
	for (r = 0; r < CNN_LAYER3_OUT_SIZE; r++){
	for (c = 0; c < CNN_LAYER3_OUT_SIZE; c++){
	PRT("%f ", pCnnNet->stCovL3.y[i][r][c]);
	}
	PRT("\n");
	}
	PRT("\n");
	}
	*/

	/* ���Ĳ��������� */
	inSize.c = pCnnNet->stPoolL4.inputWidth;
	inSize.r = pCnnNet->stPoolL4.inputHeight;
	outSize.c = inSize.c / pCnnNet->stPoolL4.mapSize;
	outSize.r = inSize.r / pCnnNet->stPoolL4.mapSize;
	/*    PRT("S4: insize[%d-%d], outsize[%d-%d]\n", inSize.c, inSize.r, outSize.c, outSize.r); */
	for (i = 0; i < (pCnnNet->stPoolL4.outChannels); i++)
	{
		if (pCnnNet->stPoolL4.poolType == AvePool)
		{
#if 0
			PoolingAverage((FLOAT *)pCnnNet->stPoolL4.y[i], outSize, (FLOAT *)pCnnNet->stCovL3.y[i], inSize, pCnnNet->stPoolL4.mapSize);
#else
			cuL4PoolingAverage((FLOAT *)pCnnNet->stPoolL4.y[i], outSize, (FLOAT *)pCnnNet->stCovL3.y[i], inSize, pCnnNet->stPoolL4.mapSize);
#endif
		}
	}

	/* �����O5�Ĵ���
	1. ������Ҫ��ǰ��Ķ�ά���չ����һά����
	*/
	pOutLayer = &pCnnNet->stOutL5;
	/*
	PRT("L4: output data \n");
	for (i = 0; i < (pCnnNet->stPoolL4.outChannels); i++){
	for (r = 0; r < outSize.r; r++){
	for (c = 0; c < outSize.c; c++){
	pCnnNet->stOutL5.inData[i*outSize.r*outSize.c + r*outSize.c + c] = pCnnNet->stPoolL4.y[i][r][c];
	PRT("%f ", pCnnNet->stPoolL4.y[i][r][c]);
	}
	}
	PRT("\n");
	}

	PRT("L5: inputNum-outputNum[%d-%d] \n", pCnnNet->stOutL5.inputNum, pCnnNet->stOutL5.outputNum);
	PRT("L5: input data \n");
	for(i = 0; i < CNN_LAYER5_IN_DATA_NUM; i++)
	{
	PRT("%f ", pCnnNet->stOutL5.inData[i]);
	if((i+1)%16 == 0)
	{
	PRT("\n");
	}
	}
	PRT("\n");
	*/
	nSize pCnnNetL5nSize = { pCnnNet->stOutL5.inputNum, pCnnNet->stOutL5.outputNum };/* (12*4*4)192-10 */
#if 0
	nn2f(pCnnNet->stOutL5.v, (FLOAT *)pCnnNet->stPoolL4.y, (FLOAT *)pCnnNet->stOutL5.wData, pCnnNet->stOutL5.basicData, pCnnNetL5nSize);
#else
	hipMemcpy((void*)(pCnnNet->stPoolL4.yDev), (void*)(pCnnNet->stPoolL4.y), 12 * 4 * 4 * sizeof(FLOAT), hipMemcpyHostToDevice);

	hipMemcpy((void*)(pOutLayer->wDev), (void*)(pOutLayer->wData), 10 * 192 * sizeof(FLOAT), hipMemcpyHostToDevice);
	hipMemcpy((void*)(pOutLayer->basicDev), (void*)(pOutLayer->basicData), 10 * sizeof(FLOAT), hipMemcpyHostToDevice);
	cuNn2f_kernel << <pOutLayer->outputNum, pOutLayer->inputNum >> >(pOutLayer->vDev, pCnnNet->stPoolL4.yDev, pOutLayer->wDev, pOutLayer->basicDev, pCnnNetL5nSize);
	hipMemcpy((void*)(pCnnNet->stOutL5.v), (void*)(pOutLayer->vDev), 10 * sizeof(FLOAT), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
#endif
	/*
	for(i = 0; i < CNN_LAYER5_OUT_CHANNEL_NUM; i++)
	{
	PRT("%f ", pCnnNet->stOutL5.v[i]);
	}
	PRT("\n");
	*/
#if 0
	for (i = 0; i < pCnnNet->stOutL5.outputNum; i++)
	{
		pCnnNet->stOutL5.y[i] = SigmaActiveFun(pCnnNet->stOutL5.v[i], pCnnNet->stOutL5.basicData[i]);
	}
#else
	INT outputDataSize = pCnnNet->stOutL5.outputNum * sizeof(FLOAT);
//	hipMemcpy((void*)(pOutLayer->vDev), (void*)(pCnnNet->stOutL5.v), outputDataSize, hipMemcpyHostToDevice);
//	hipMemcpy((void*)(pOutLayer->basicDev), (void*)(pCnnNet->stOutL5.basicData), outputDataSize, hipMemcpyHostToDevice);
	cuSigmaActiveL5_kernel << <1, CNN_LAYER5_OUT_CHANNEL_NUM >> >(pOutLayer->vDev, pOutLayer->basicDev, pOutLayer->yDev);
	hipDeviceSynchronize();
	hipMemcpy((void*)(pCnnNet->stOutL5.y), (void*)(pOutLayer->yDev), outputDataSize, hipMemcpyDeviceToHost);
#endif
}

