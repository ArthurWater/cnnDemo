#include "hip/hip_runtime.h"
/*

meimaokui@126.com
*/


#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "inc/com_type_def.h"
#include "inc/common.h"
#include "inc/cnn.h"
#include "inc/minst.h"
#include "inc/cnn_inference.h"
#include "inc/cuCnn.h"

void GetGpuDeviceInfo();

//global variable 
CNN_NET_STR stCnn;
CNN_NET_STR stCnnTrain;



int main()
{
	int i = 0;
	int j = 0;
	int k = 0;
	int idx = 0;
	int iRet = 0;
	FILE *fp = NULL;
	int maxIndex = 0;
	int iLabelIndex = 0;
	int incorrectnum = 0;  /* ����Ԥ�����Ŀ */
	CHAR imgName[128] = { '\0' };
	UNCHAR imgData[CNN_LAYER1_IN_SIZE][CNN_LAYER1_IN_SIZE] = { { 0 } };/* �Ҷ�ͼ */
	FLOAT imgDataFloat[CNN_LAYER1_IN_SIZE][CNN_LAYER1_IN_SIZE] = { { 0.0 } };/* �Ҷ�ͼ */
	MinstImgArr testImg;
	MinstLabelArr testLabel;
	
	/* CNN�ṹ�ĳ�ʼ�� */
	memset(&stCnn, 0, sizeof(stCnn));
	memset(&stCnnTrain, 0, sizeof(stCnnTrain));
	memset(&testImg, 0, sizeof(MinstImgArr));
	memset(&testLabel, 0, sizeof(MinstLabelArr));

	//���GPU��Ϣ
	GetGpuDeviceInfo();
	PRT("LeNet-5 run ......\n");
	minstReadLable(&testLabel, CNN_TEST_LABELS_PATH);
	ReadMinstImg(&testImg, CNN_TEST_IMAGES_PATH);

	nSize inputSize = { testImg.ImgPtr[0].c, testImg.ImgPtr[0].r };
	INT32 outSize = testLabel.LabelPtr[0].len;

	PRT("input c:%d, r:%d, outSize:%d\n", inputSize.c, inputSize.r, outSize);/* 28,28,10 */
	PRT("ImgNum:%d\n", testImg.ImgNum);

	PRT("loadind model file ...\n");
#if 0
	CnnSetUp(&stCnn);
#else
	cuCnnSetUp(&stCnn);
#endif

	iRet = ImportCnnModelFile(&stCnn, CNN_MODEL_FILE_SAVE_PATH);
	if (iRet)
	{
		PRT_ERR("import cnn error !\n");
		system("pause");
		return -1;
	}

	for (idx = 0; idx < PIC_TEST_NUM; idx++)
	{
		sprintf(imgName, "./output/img_%d.yuv", idx);
		fp = fopen(imgName, "wb");
		for (j = 0; j < CNN_LAYER1_IN_SIZE; j++)
		{
			for (k = 0; k < CNN_LAYER1_IN_SIZE; k++)
			{
				imgDataFloat[j][k] = testImg.ImgPtr[idx].ImgData[j][k];
				imgData[j][k] = (unsigned char)(testImg.ImgPtr[idx].ImgData[j][k] * 255);
			}
		}

		fwrite(imgData, CNN_LAYER1_IN_SIZE * CNN_LAYER1_IN_SIZE, 1, fp);
		fclose(fp);
#if 0
		CnnForwardPass(&stCnn, (FLOAT *)imgDataFloat);
#else
		cuCnnForwardPass(&stCnn, (FLOAT *)imgDataFloat);
#endif
		maxIndex = vecMaxIndex(stCnn.stOutL5.y, stCnn.stOutL5.outputNum);
		iLabelIndex = vecMaxIndex(testLabel.LabelPtr[idx].LabelData, stCnn.stOutL5.outputNum);

		/*
		PRT("Test:");
		for(i = 0; i < CNN_LAYER5_OUT_CHANNEL_NUM; i++)
		{
		PRT("[%d]:%f ", i, stCnn.stOutL5.y[i]);
		}
		PRT("\n");

		PRT("Real:");
		for(i = 0; i < CNN_LAYER5_OUT_CHANNEL_NUM; i++)
		{
		PRT("[%d]:%f ", i, testLabel.LabelPtr[n].LabelData[i]);
		}
		PRT("\n");
		*/
		if (maxIndex != iLabelIndex)
		{
			incorrectnum++;
		}

		PRT("outputNum:%d, testIndex:%d, realIndex:%d\n", stCnn.stOutL5.outputNum, maxIndex, iLabelIndex);

		CnnParamClear(&stCnn);
	}

	PRT("incorrectnum/totalNUmbet: %d/%d\n", incorrectnum, PIC_TEST_NUM);
	cuCnnDestroy(&stCnn);
	system("pause");
    return 0;
}




void GetGpuDeviceInfo()
{
	int deviceCount = 0;
	int dev = 0;
	int driverVersion = 0; 
	int runtimeVersion = 0;
	hipDeviceProp_t deviceProp;

	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n",(int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	if (deviceCount == 0) {
		PRT("There are no available device(s) that support CUDA\n");
	}
	else {
		PRT("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	hipSetDevice(dev);
	hipGetDeviceProperties(&deviceProp, dev);
	PRT("Device %d: \"%s\"\n", dev, deviceProp.name);
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	PRT(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
	PRT(" CUDA Capability Major/Minor version number: %d.%d\n", deviceProp.major, deviceProp.minor);
	PRT(" Total amount of global memory: %.2f GBytes (%llu bytes)\n", (float)deviceProp.totalGlobalMem / (pow(1024.0, 3)), (unsigned long long) deviceProp.totalGlobalMem);
	PRT(" GPU Clock rate: %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
	PRT(" Memory Clock rate: %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
	PRT(" Memory Bus Width: %d-bit\n", deviceProp.memoryBusWidth);

	if (deviceProp.l2CacheSize) {
		PRT(" L2 Cache Size: %d bytes\n",
			deviceProp.l2CacheSize);
	}

	PRT(" Max Texture Dimension Size (x,y,z) 1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
		deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
		deviceProp.maxTexture2D[1],
		deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
		deviceProp.maxTexture3D[2]);

	PRT(" Max Layered Texture Size (dim) x layers 1D=(%d) x %d, 2D=(%d,%d) x %d\n",
		deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
		deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
		deviceProp.maxTexture2DLayered[2]);

	PRT(" Total amount of constant memory: %lu bytes\n", deviceProp.totalConstMem);
	PRT(" Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
	PRT(" Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
	PRT(" Warp size: %d\n", deviceProp.warpSize);
	PRT(" Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	PRT(" Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	PRT(" Maximum sizes of each dimension of a block: %d x %d x %d\n",
		deviceProp.maxThreadsDim[0],
		deviceProp.maxThreadsDim[1],
		deviceProp.maxThreadsDim[2]);
	PRT(" Maximum sizes of each dimension of a grid: %d x %d x %d\n",
		deviceProp.maxGridSize[0],
		deviceProp.maxGridSize[1],
		deviceProp.maxGridSize[2]);

	PRT(" Maximum memory pitch: %lu bytes\n", deviceProp.memPitch);
}

